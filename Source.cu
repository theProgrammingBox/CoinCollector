#include "Network.cuh"

#define BOARD_WIDTH 4
#define BOARD_SIZE (BOARD_WIDTH * BOARD_WIDTH)
#define VIS_SIZE (BOARD_SIZE - 1)
#define ACTIONS 4
#define INPUTS (BOARD_SIZE + 1)
#define SCORE_SIZE 1000

#define QUEUE_SIZE 16384
#define MIN_QUEUE_SIZE 2048
#define BATCH_SIZE 1024
#define LEARNING_RATE 0.001f
#define WEIGHT_DECAY 0.00f
#define REWARD_DECAY 0.99f
#define EPOCHES 40000

int main(int argc, char **argv) {
    Noise noise;
    initNoise(&noise);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    Network net;
    uint32_t parameters[] = {INPUTS, 16, 16, ACTIONS};
    uint32_t layers = sizeof(parameters) / sizeof(uint32_t) - 1;
    initNetwork(&net, parameters, layers, &noise, LEARNING_RATE, BATCH_SIZE > VIS_SIZE ? BATCH_SIZE : VIS_SIZE, WEIGHT_DECAY);
    
    float states[BOARD_SIZE * QUEUE_SIZE];
    uint8_t actions[QUEUE_SIZE];
    float rewards[QUEUE_SIZE];
    float nextStates[BOARD_SIZE * QUEUE_SIZE];
    uint32_t queueIdx = 0;
    
    uint32_t sampledIdxs[BATCH_SIZE];
    float outputs[ACTIONS * (BATCH_SIZE > VIS_SIZE ? BATCH_SIZE : VIS_SIZE)];
    float outputGrads[ACTIONS * BATCH_SIZE];
    float bestScores[BATCH_SIZE];
    
    float board[BOARD_SIZE]{};
    float score[SCORE_SIZE]{};
    uint8_t px, py, cx, cy;
    uint32_t scoreIdx = 0;
    float scoreSum = 0.0f;
    
    px = genNoise(&noise) % BOARD_WIDTH;
    py = genNoise(&noise) % BOARD_WIDTH;
    do {
        cx = genNoise(&noise) % BOARD_WIDTH;
        cy = genNoise(&noise) % BOARD_WIDTH;
    } while (cx == px && cy == py);
    board[py * BOARD_WIDTH + px] = 1.0f;
    board[cy * BOARD_WIDTH + cx] = -1.0f;
    
    const float one = 1.0f;
    for (uint32_t epoch = 0; epoch < EPOCHES; epoch++) {
        memcpy(states + queueIdx * BOARD_SIZE, board, BOARD_SIZE * sizeof(float));
        
        
        uint8_t action;
        net.batchSize = VIS_SIZE;
        // hipMemcpy(net.outputs[0], board, BOARD_SIZE * sizeof(float), hipMemcpyHostToDevice);
        // hipMemcpy(net.outputs[0] + BOARD_SIZE, &one, sizeof(float), hipMemcpyHostToDevice);
        board[py * BOARD_WIDTH + px] = 0.0f;
        uint32_t idx = 0;
        for (uint8_t pyy = 0; pyy < BOARD_WIDTH; pyy++) {
            for (uint8_t pxx = 0; pxx < BOARD_WIDTH; pxx++) {
                if (pxx == cx && pyy == cy) continue;
                board[pyy * BOARD_WIDTH + pxx] = 1.0f;
                hipMemcpy(net.outputs[0] + idx * INPUTS, board, BOARD_SIZE * sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(net.outputs[0] + idx * INPUTS + BOARD_SIZE, &one, sizeof(float), hipMemcpyHostToDevice);
                idx++;
                board[pyy * BOARD_WIDTH + pxx] = 0.0f;
            }
        }
        board[py * BOARD_WIDTH + px] = 1.0f;
        // forwardNoisy(&handle, &net, &noise);
        forwardNoiseless(&handle, &net);
        // hipMemcpy(outputs, net.outputs[net.layers], ACTIONS * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(outputs, net.outputs[net.layers], ACTIONS * VIS_SIZE * sizeof(float), hipMemcpyDeviceToHost);
        if (genNoise(&noise) % 64 < (epoch < EPOCHES * 0.1 ? 64 : 0)) {
            action = genNoise(&noise) % ACTIONS;
        } else {
            action = 0;
            uint32_t pos = py * BOARD_WIDTH + px;
            uint8_t bias = pos > (cy * BOARD_WIDTH + cx);
            for (uint8_t i = 1; i < ACTIONS; i++) {
                if (outputs[i + (pos - bias) * ACTIONS] > outputs[action + (pos - bias) * ACTIONS]) {
                    action = i;
                }
            }
        }
        
        float maxScore = outputs[0];
        float minScore = outputs[0];
        for (uint8_t i = 1; i < ACTIONS * VIS_SIZE; i++) {
            if (outputs[i] > maxScore) {
                maxScore = outputs[i];
            }
            if (outputs[i] < minScore) {
                minScore = outputs[i];
            }
        }
        printf("\033[H\033[J");
        printf("%d/%d\n", epoch, EPOCHES);
        idx = 0;
        for (uint8_t y = 0; y < BOARD_WIDTH; y++) {
            for (uint8_t x = 0; x < BOARD_WIDTH; x++) {
                if (x == cx && y == cy) {
                    printf("\x1b[38;2;255;255;0m");
                    printf("$$");
                } else {
                    uint8_t act = 0;
                    float bestScore = outputs[idx * ACTIONS];
                    for (uint8_t i = 1; i < ACTIONS; i++) {
                        if (outputs[idx * ACTIONS + i] > bestScore) {
                            bestScore = outputs[idx * ACTIONS + i];
                            act = i;
                        }
                    }
                    if (x == px && y == py) {
                        printf("\x1b[38;2;255;0;255m");
                    } else {
                        uint8_t g = (bestScore - minScore) / (maxScore - minScore) * 255;
                        printf("\x1b[38;2;%d;%d;0m", 255 - g, g);
                    }
                    switch (act) {
                        case 0: printf("<<"); break;
                        case 1: printf(">>"); break;
                        case 2: printf("^^"); break;
                        case 3: printf("vv"); break;
                    }
                    idx++;
                }
            }
            printf("\n");
        }
        printf("\x1b[38;2;255;255;255m");
        
        board[py * BOARD_WIDTH + px] = 0.0f;
        switch (action) {
            case 0: if (px > 0) px--; break;
            case 1: if (px < BOARD_WIDTH - 1) px++; break;
            case 2: if (py > 0) py--; break;
            case 3: if (py < BOARD_WIDTH - 1) py++; break;
        }
        board[py * BOARD_WIDTH + px] = 1.0f;
        float reward = cx == px && cy == py;
        
        actions[queueIdx] = action;
        rewards[queueIdx] = reward;
        
        while (cx == px && cy == py) {
            cx = genNoise(&noise) % BOARD_WIDTH;
            cy = genNoise(&noise) % BOARD_WIDTH;
        }
        board[cy * BOARD_WIDTH + cx] = -1.0f;
        memcpy(nextStates + queueIdx * BOARD_SIZE, board, BOARD_SIZE * sizeof(float));
        queueIdx *= ++queueIdx != QUEUE_SIZE;
        
        score[scoreIdx] = reward;
        scoreSum += reward;
        scoreIdx *= ++scoreIdx != SCORE_SIZE;
        scoreSum -= score[scoreIdx];
        uint32_t scoreIdxCap = epoch >= SCORE_SIZE ? SCORE_SIZE : epoch + 1;
        printf("Average score: %f\n", scoreSum / scoreIdxCap);
        if (epoch > EPOCHES * 0.9 && scoreSum / scoreIdxCap > 0.31f) {
            struct timeval tv;
            tv.tv_sec = 0;
            tv.tv_usec = 1000000;
            select(0, NULL, NULL, NULL, &tv);
        }
        // printTensor(net.weightSamples[0], net.parameters[1], net.parameters[0]);
        // printTensor(net.weightVars[0], net.parameters[1], net.parameters[0]);
        // printTensor(net.weightMeans[0], net.parameters[1], net.parameters[0]);
        
        
        if (epoch + 1 < MIN_QUEUE_SIZE) continue;
        uint32_t idxCap = epoch >= QUEUE_SIZE ? QUEUE_SIZE : epoch;
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            sampledIdxs[i] = genNoise(&noise) % idxCap;
        }
        
        net.batchSize = BATCH_SIZE;
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            hipMemcpy(net.outputs[0] + i * INPUTS, nextStates + sampledIdxs[i] * BOARD_SIZE, BOARD_SIZE * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(net.outputs[0] + i * INPUTS + BOARD_SIZE, &one, sizeof(float), hipMemcpyHostToDevice);
        }
        forwardNoiseless(&handle, &net);
        hipMemcpy(outputs, net.outputs[net.layers], ACTIONS * BATCH_SIZE * sizeof(float), hipMemcpyDeviceToHost);
        
        float bestScore;
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            bestScore = outputs[i * ACTIONS];
            for (uint8_t j = 1; j < ACTIONS; j++) {
                if (outputs[i * ACTIONS + j] > bestScore) {
                    bestScore = outputs[i * ACTIONS + j];
                }
            }
            bestScores[i] = bestScore;
        }
        
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            hipMemcpy(net.outputs[0] + i * INPUTS, states + sampledIdxs[i] * BOARD_SIZE, BOARD_SIZE * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(net.outputs[0] + i * INPUTS + BOARD_SIZE, &one, sizeof(float), hipMemcpyHostToDevice);
        }
        // forwardNoisy(&handle, &net, &noise);
        forwardNoiseless(&handle, &net);
        hipMemcpy(outputs, net.outputs[net.layers], ACTIONS * BATCH_SIZE * sizeof(float), hipMemcpyDeviceToHost);
        memset(outputGrads, 0, ACTIONS * BATCH_SIZE * sizeof(float));
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            outputGrads[i * ACTIONS + actions[sampledIdxs[i]]] = rewards[sampledIdxs[i]] + REWARD_DECAY * bestScores[i] - outputs[i * ACTIONS + actions[sampledIdxs[i]]];
        }
        hipMemcpy(net.outputGrads[net.layers], outputGrads, ACTIONS * BATCH_SIZE * sizeof(float), hipMemcpyHostToDevice);
        // backwardNoisy(&handle, &net);
        backwardNoiseless(&handle, &net);
    }

    return 0;
}