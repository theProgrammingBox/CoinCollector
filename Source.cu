#include "Network.cuh"

#define BOARD_WIDTH 4
#define BOARD_SIZE (BOARD_WIDTH * BOARD_WIDTH)
#define VIS_SIZE (BOARD_SIZE - 1)
#define ACTIONS 4
#define INPUTS (BOARD_SIZE + 1)
#define SCORE_SIZE 1000

#define QUEUE_SIZE 32768
#define MIN_QUEUE_SIZE 4096
#define BATCH_SIZE 128 // 32
#define LEARNING_RATE 0.001f
#define WEIGHT_DECAY 0.00f
#define REWARD_DECAY 0.99f
#define EPOCHES 40000

int main(int argc, char **argv) {
    Noise noise;
    initNoise(&noise);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    Network net;
    uint32_t parameters[] = {INPUTS, 16, 16, ACTIONS};
    uint32_t layers = sizeof(parameters) / sizeof(uint32_t) - 1;
    initNetwork(&net, parameters, layers, &noise, LEARNING_RATE, BATCH_SIZE > VIS_SIZE ? BATCH_SIZE : VIS_SIZE, WEIGHT_DECAY);
    
    float states[BOARD_SIZE * QUEUE_SIZE];
    uint8_t actions[QUEUE_SIZE];
    float rewards[QUEUE_SIZE];
    float nextStates[BOARD_SIZE * QUEUE_SIZE];
    uint32_t queueIdx = 0;
    
    uint32_t sampledIdxs[BATCH_SIZE];
    float outputs[ACTIONS * (BATCH_SIZE > VIS_SIZE ? BATCH_SIZE : VIS_SIZE)];
    float outputGrads[ACTIONS * BATCH_SIZE];
    float bestScores[BATCH_SIZE];
    
    float board[BOARD_SIZE]{};
    float score[SCORE_SIZE]{};
    uint8_t px, py, cx, cy;
    uint32_t scoreIdx = 0;
    float scoreSum = 0.0f;
    
    px = genUint(&noise) % BOARD_WIDTH;
    py = genUint(&noise) % BOARD_WIDTH;
    do {
        cx = genUint(&noise) % BOARD_WIDTH;
        cy = genUint(&noise) % BOARD_WIDTH;
    } while (cx == px && cy == py);
    board[py * BOARD_WIDTH + px] = 1.0f;
    board[cy * BOARD_WIDTH + cx] = -1.0f;
    
    const float one = 1.0f;
    for (uint32_t epoch = 0; epoch < EPOCHES; epoch++) {
        memcpy(states + queueIdx * BOARD_SIZE, board, BOARD_SIZE * sizeof(float));
        
        
        uint8_t action;
        net.batchSize = VIS_SIZE;
        // hipMemcpy(net.outputs[0], board, BOARD_SIZE * sizeof(float), hipMemcpyHostToDevice);
        // hipMemcpy(net.outputs[0] + BOARD_SIZE, &one, sizeof(float), hipMemcpyHostToDevice);
        board[py * BOARD_WIDTH + px] = 0.0f;
        uint32_t idx = 0;
        for (uint8_t pyy = 0; pyy < BOARD_WIDTH; pyy++) {
            for (uint8_t pxx = 0; pxx < BOARD_WIDTH; pxx++) {
                if (pxx == cx && pyy == cy) continue;
                board[pyy * BOARD_WIDTH + pxx] = 1.0f;
                hipMemcpy(net.outputs[0] + idx * INPUTS, board, BOARD_SIZE * sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(net.outputs[0] + idx * INPUTS + BOARD_SIZE, &one, sizeof(float), hipMemcpyHostToDevice);
                idx++;
                board[pyy * BOARD_WIDTH + pxx] = 0.0f;
            }
        }
        board[py * BOARD_WIDTH + px] = 1.0f;
        // forwardNoisy(&handle, &net, &noise);
        forwardNoiseless(&handle, &net);
        // hipMemcpy(outputs, net.outputs[net.layers], ACTIONS * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(outputs, net.outputs[net.layers], ACTIONS * VIS_SIZE * sizeof(float), hipMemcpyDeviceToHost);
        float epsilon = (epoch / (EPOCHES * 0.9f));
        epsilon = epsilon > 1.0f ? 0.0f : 1 - epsilon;
        action = 0;
        uint32_t pos = py * BOARD_WIDTH + px;
        uint8_t bias = pos > (cy * BOARD_WIDTH + cx);
        float bestScore = outputs[(pos - bias) * ACTIONS] + genNormal(&noise) * epsilon;
        // float bestScore = outputs[(pos - bias) * ACTIONS];
        for (uint8_t i = 1; i < ACTIONS; i++) {
            float sample = outputs[(pos - bias) * ACTIONS + i] + genNormal(&noise) * epsilon;
            // float sample = outputs[(pos - bias) * ACTIONS + i];
            if (sample > bestScore) {
                bestScore = sample;
                action = i;
            }
        }
        
        float maxScore = -INFINITY;
        float minScore = INFINITY;
        for (uint8_t i = 1; i < VIS_SIZE; i++) {
            float bestScore = outputs[i * ACTIONS];
            for (uint8_t j = 1; j < ACTIONS; j++) {
                if (outputs[i * ACTIONS + j] > bestScore) {
                    bestScore = outputs[i * ACTIONS + j];
                }
            }
            if (bestScore > maxScore) {
                maxScore = bestScore;
            }
            if (bestScore < minScore) {
                minScore = bestScore;
            }
        }
        printf("\033[H\033[J");
        printf("%d/%d\n", epoch, EPOCHES);
        idx = 0;
        for (uint8_t y = 0; y < BOARD_WIDTH; y++) {
            for (uint8_t x = 0; x < BOARD_WIDTH; x++) {
                if (x == cx && y == cy) {
                    printf("\x1b[38;2;255;255;0m");
                    printf("$$");
                } else {
                    uint8_t act = 0;
                    float bestScore = outputs[idx * ACTIONS];
                    for (uint8_t i = 1; i < ACTIONS; i++) {
                        if (outputs[idx * ACTIONS + i] > bestScore) {
                            bestScore = outputs[idx * ACTIONS + i];
                            act = i;
                        }
                    }
                    if (x == px && y == py) {
                        printf("\x1b[38;2;255;0;255m");
                    } else {
                        uint8_t g = (bestScore - minScore) / (maxScore - minScore) * 255;
                        printf("\x1b[38;2;%d;%d;0m", 255 - g, g);
                    }
                    switch (act) {
                        case 0: printf("<<"); break;
                        case 1: printf(">>"); break;
                        case 2: printf("^^"); break;
                        case 3: printf("vv"); break;
                    }
                    idx++;
                }
            }
            printf("\n");
        }
        printf("\x1b[38;2;255;255;255m");
        
        board[py * BOARD_WIDTH + px] = 0.0f;
        switch (action) {
            case 0: if (px > 0) px--; break;
            case 1: if (px < BOARD_WIDTH - 1) px++; break;
            case 2: if (py > 0) py--; break;
            case 3: if (py < BOARD_WIDTH - 1) py++; break;
        }
        board[py * BOARD_WIDTH + px] = 1.0f;
        float reward = cx == px && cy == py;
        
        actions[queueIdx] = action;
        rewards[queueIdx] = reward;
        
        while (cx == px && cy == py) {
            cx = genUint(&noise) % BOARD_WIDTH;
            cy = genUint(&noise) % BOARD_WIDTH;
        }
        board[cy * BOARD_WIDTH + cx] = -1.0f;
        memcpy(nextStates + queueIdx * BOARD_SIZE, board, BOARD_SIZE * sizeof(float));
        queueIdx *= ++queueIdx != QUEUE_SIZE;
        
        score[scoreIdx] = reward;
        scoreSum += reward;
        scoreIdx *= ++scoreIdx != SCORE_SIZE;
        scoreSum -= score[scoreIdx];
        uint32_t scoreIdxCap = epoch >= SCORE_SIZE ? SCORE_SIZE : epoch + 1;
        printf("Average score: %f\n", scoreSum / scoreIdxCap);
        if (epoch > EPOCHES * 0.9 && scoreSum / scoreIdxCap > 0.31f) {
            struct timeval tv;
            tv.tv_sec = 0;
            tv.tv_usec = 1000000;
            select(0, NULL, NULL, NULL, &tv);
        }
        // printTensor(net.weightSamples[0], net.parameters[1], net.parameters[0]);
        // printTensor(net.weightVars[0], net.parameters[1], net.parameters[0]);
        // printTensor(net.weightMeans[0], net.parameters[1], net.parameters[0]);
        
        
        if (epoch + 1 < MIN_QUEUE_SIZE) continue;
        uint32_t idxCap = epoch + 1 >= QUEUE_SIZE ? QUEUE_SIZE : epoch + 1;
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            sampledIdxs[i] = genUint(&noise) % idxCap;
        }
        
        net.batchSize = BATCH_SIZE;
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            hipMemcpy(net.outputs[0] + i * INPUTS, nextStates + sampledIdxs[i] * BOARD_SIZE, BOARD_SIZE * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(net.outputs[0] + i * INPUTS + BOARD_SIZE, &one, sizeof(float), hipMemcpyHostToDevice);
        }
        forwardNoiseless(&handle, &net);
        // forwardNoisy(&handle, &net, &noise);
        hipMemcpy(outputs, net.outputs[net.layers], ACTIONS * BATCH_SIZE * sizeof(float), hipMemcpyDeviceToHost);
        
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            float bestScore = outputs[i * ACTIONS];
            for (uint8_t j = 1; j < ACTIONS; j++) {
                if (outputs[i * ACTIONS + j] > bestScore) {
                    bestScore = outputs[i * ACTIONS + j];
                }
            }
            bestScores[i] = bestScore;
        }
        
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            hipMemcpy(net.outputs[0] + i * INPUTS, states + sampledIdxs[i] * BOARD_SIZE, BOARD_SIZE * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(net.outputs[0] + i * INPUTS + BOARD_SIZE, &one, sizeof(float), hipMemcpyHostToDevice);
        }
        // forwardNoisy(&handle, &net, &noise);
        forwardNoiseless(&handle, &net);
        hipMemcpy(outputs, net.outputs[net.layers], ACTIONS * BATCH_SIZE * sizeof(float), hipMemcpyDeviceToHost);
        memset(outputGrads, 0, ACTIONS * BATCH_SIZE * sizeof(float));
        for (uint32_t i = 0; i < BATCH_SIZE; i++) {
            outputGrads[i * ACTIONS + actions[sampledIdxs[i]]] = rewards[sampledIdxs[i]] + REWARD_DECAY * bestScores[i] - outputs[i * ACTIONS + actions[sampledIdxs[i]]];
        }
        hipMemcpy(net.outputGrads[net.layers], outputGrads, ACTIONS * BATCH_SIZE * sizeof(float), hipMemcpyHostToDevice);
        // backwardNoisy(&handle, &net);
        backwardNoiseless(&handle, &net);
    }

    return 0;
}