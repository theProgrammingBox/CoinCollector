
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>

#include <hipblas.h>

#define BOARD_WIDTH 3
#define BOARD_SIZE (BOARD_WIDTH * BOARD_WIDTH)
#define ACTIONS 4
#define NUM_FINAL_STATES (BOARD_SIZE * (BOARD_SIZE - 1) * ACTIONS)

#define HIDDEN_LAYER_SIZE 16
#define DECAY 0.9

uint32_t mixSeed(uint32_t *seed1, uint32_t *seed2) {
    *seed2 ^= (*seed1 >> 17) * 0xbf324c81;
    *seed1 ^= (*seed2 >> 13) * 0x9c7493ad;
    return *seed1;
}

void initializeSeeds(uint32_t *seed1, uint32_t *seed2) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    *seed1 = tv.tv_sec;
    *seed2 = tv.tv_usec;
    for (uint8_t i = 8; i--;) mixSeed(seed1, seed2);
}

__global__ void _fillUniform(float* arr, uint32_t size, uint32_t seed1, uint32_t seed2, float lowerBound, float upperBound) {
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        uint32_t hash = index;
        hash ^= (hash ^ seed1) * 0x4ba1bb47;
        hash ^= (hash ^ seed2) * 0xb7ebcb79;
        hash ^= hash << 5;
        arr[index] = hash * 0.00000000023283064365386962890625f * (upperBound - lowerBound) + lowerBound;
    }
}

void fillUniform(float* arr, uint32_t size, uint32_t* seed1, uint32_t* seed2, float lowerBound, float upperBound) {
    mixSeed(seed1, seed2);
    _fillUniform<<<(size >> 10) + (size & 0x3ff ? 1 : 0), 1024>>>(arr, size, *seed1, *seed2, lowerBound, upperBound);
}

__global__ void _reluForward(float *dTensor, uint32_t size) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
    dTensor[idx] = dTensor[idx] > 0 ? dTensor[idx] : 0;
}

void reluForward(float *dTensor, uint32_t size) {
    _reluForward<<<(size >> 10) + (size & 0x3ff ? 1 : 0), 0x400>>>(dTensor, size);
}

__global__ void _reluBackward(float *dTensor, float *dTensorGrad, uint32_t size) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
    dTensorGrad[idx] = dTensor[idx] > 0 ? dTensorGrad[idx] : 0;
}

void reluBackward(float *dTensor, float *dTensorGrad, uint32_t size) {
    _reluBackward<<<(size >> 10) + (size & 0x3ff), 0x400>>>(dTensor, dTensorGrad, size);
}

struct Model {
    float* weight1;//[BOARD_SIZE * HIDDEN_LAYER_SIZE];
    float* weight2;//[HIDDEN_LAYER_SIZE * ACTIONS];
    float* bias1;//[HIDDEN_LAYER_SIZE];
    float* bias2;//[ACTIONS];
    
    float* input;//[BOARD_SIZE * NUM_FINAL_STATES];
    float* hidden;//[HIDDEN_LAYER_SIZE * NUM_FINAL_STATES];
    float* output;//[ACTIONS * NUM_FINAL_STATES];
    
    float* weight1Grad;//[BOARD_SIZE * HIDDEN_LAYER_SIZE];
    float* weight2Grad;//[HIDDEN_LAYER_SIZE * ACTIONS];
    
    float* hiddenGrad;//[HIDDEN_LAYER_SIZE * NUM_FINAL_STATES];
    float* outputGrad;//[BOARD_SIZE * NUM_FINAL_STATES];
};

void initializeModel(Model *model, uint32_t* seed1, uint32_t* seed2) {
    hipMalloc((void**)&model->weight1, BOARD_SIZE * HIDDEN_LAYER_SIZE * sizeof(float));
    hipMalloc((void**)&model->weight2, HIDDEN_LAYER_SIZE * ACTIONS * sizeof(float));
    hipMalloc((void**)&model->bias1, HIDDEN_LAYER_SIZE * sizeof(float));
    hipMalloc((void**)&model->bias2, ACTIONS * sizeof(float));
    
    hipMalloc((void**)&model->input, BOARD_SIZE * NUM_FINAL_STATES * sizeof(float));
    hipMalloc((void**)&model->hidden, HIDDEN_LAYER_SIZE * NUM_FINAL_STATES * sizeof(float));
    hipMalloc((void**)&model->output, ACTIONS * NUM_FINAL_STATES * sizeof(float));
    
    hipMalloc((void**)&model->weight1Grad, BOARD_SIZE * HIDDEN_LAYER_SIZE * sizeof(float));
    hipMalloc((void**)&model->weight2Grad, HIDDEN_LAYER_SIZE * ACTIONS * sizeof(float));
    
    hipMalloc((void**)&model->hiddenGrad, HIDDEN_LAYER_SIZE * NUM_FINAL_STATES * sizeof(float));
    hipMalloc((void**)&model->outputGrad, BOARD_SIZE * NUM_FINAL_STATES * sizeof(float));
    
    fillUniform(model->weight1, BOARD_SIZE * HIDDEN_LAYER_SIZE, seed1, seed2, -0.1, 0.1);
    fillUniform(model->weight2, HIDDEN_LAYER_SIZE * ACTIONS, seed1, seed2, -0.1, 0.1);
    fillUniform(model->bias1, HIDDEN_LAYER_SIZE, seed1, seed2, -0.1, 0.1);
    fillUniform(model->bias2, ACTIONS, seed1, seed2, -0.1, 0.1);
}

void forward(hipblasHandle_t* handle, Model *model) {
    const float ONE = 1;
    
    for (uint32_t i = 0; i < NUM_FINAL_STATES; i++) {
        hipMemcpy(model->hidden + i * HIDDEN_LAYER_SIZE, model->bias1, HIDDEN_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToDevice);
    }
    
    hipblasSgemm(
        *handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        HIDDEN_LAYER_SIZE, NUM_FINAL_STATES, BOARD_SIZE,
        &ONE,
        model->weight1, HIDDEN_LAYER_SIZE,
        model->input, BOARD_SIZE,
        &ONE,
        model->hidden, HIDDEN_LAYER_SIZE
    );
    
    reluForward(model->hidden, HIDDEN_LAYER_SIZE * NUM_FINAL_STATES);
    for (uint32_t i = 0; i < NUM_FINAL_STATES; i++) {
        hipMemcpy(model->output + i * ACTIONS, model->bias2, ACTIONS * sizeof(float), hipMemcpyDeviceToDevice);
    }
    hipblasSgemm(
        *handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        ACTIONS, NUM_FINAL_STATES, HIDDEN_LAYER_SIZE,
        &ONE,
        model->weight2, ACTIONS,
        model->hidden, HIDDEN_LAYER_SIZE,
        &ONE,
        model->output, ACTIONS
    );
}

void backward(hipblasHandle_t* handle, Model *model) {
    const float learingRate = 0.00016;
    const float ONE = 1;
    const float ZERO = 0;
    
    hipblasSgemm(
        *handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
        ACTIONS, HIDDEN_LAYER_SIZE, NUM_FINAL_STATES,
        &ONE,
        model->outputGrad, ACTIONS,
        model->hidden, HIDDEN_LAYER_SIZE,
        &ZERO,
        model->weight2Grad, ACTIONS
    );
    
    hipblasSgemm(
        *handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
        HIDDEN_LAYER_SIZE, NUM_FINAL_STATES, ACTIONS,
        &ONE,
        model->weight2, ACTIONS,
        model->outputGrad, ACTIONS,
        &ZERO,
        model->hiddenGrad, HIDDEN_LAYER_SIZE
    );
    
    reluBackward(model->hidden, model->hiddenGrad, HIDDEN_LAYER_SIZE * NUM_FINAL_STATES);
    
    hipblasSgemm(
        *handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
        HIDDEN_LAYER_SIZE, BOARD_SIZE, NUM_FINAL_STATES,
        &ONE,
        model->hiddenGrad, HIDDEN_LAYER_SIZE,
        model->input, BOARD_SIZE,
        &ZERO,
        model->weight1Grad, HIDDEN_LAYER_SIZE
    );
    
    hipblasSaxpy(*handle, HIDDEN_LAYER_SIZE * ACTIONS, &learingRate, model->weight2Grad, 1, model->weight2, 1);
    hipblasSaxpy(*handle, BOARD_SIZE * HIDDEN_LAYER_SIZE, &learingRate, model->weight1Grad, 1, model->weight1, 1);
    hipblasSaxpy(*handle, ACTIONS, &learingRate, model->outputGrad, 1, model->bias2, 1);
    hipblasSaxpy(*handle, HIDDEN_LAYER_SIZE, &learingRate, model->hiddenGrad, 1, model->bias1, 1);
}

void copyParams(Model *model, Model *frozenModel) {
    hipMemcpy(frozenModel->weight1, model->weight1, BOARD_SIZE * HIDDEN_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(frozenModel->weight2, model->weight2, HIDDEN_LAYER_SIZE * ACTIONS * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(frozenModel->bias1, model->bias1, HIDDEN_LAYER_SIZE * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(frozenModel->bias2, model->bias2, ACTIONS * sizeof(float), hipMemcpyDeviceToDevice);
}

int main(int argc, char *argv[])
{
    uint32_t seed1, seed2;
    initializeSeeds(&seed1, &seed2);
    
    float states[BOARD_SIZE * NUM_FINAL_STATES];
    uint8_t actions[NUM_FINAL_STATES];
    float rewards[NUM_FINAL_STATES];
    float nextStates[BOARD_SIZE * NUM_FINAL_STATES];
    
    uint32_t queueIdx = 0;
    float board[9]{};
    for (uint8_t py = 0; py < BOARD_WIDTH; py++) {
        for (uint8_t px = 0; px < BOARD_WIDTH; px++) {
            for (uint8_t cy = 0; cy < BOARD_WIDTH; cy++) {
                for (uint8_t cx = 0; cx < BOARD_WIDTH; cx++) {
                    if (px == cx && py == cy) continue;
                    board[py * BOARD_WIDTH + px] = 1;
                    board[cy * BOARD_WIDTH + cx] = -1;
                    for (uint8_t a = 0; a < ACTIONS; a++) {
                        memcpy(states + queueIdx * ACTIONS * BOARD_SIZE + a * BOARD_SIZE, board, BOARD_SIZE * sizeof(float));
                        uint8_t pxx = px;
                        uint8_t pyy = py;
                        switch (a) {
                            case 0: if (pxx > 0) pxx--; break;
                            case 1: if (pxx < BOARD_WIDTH - 1) pxx++; break;
                            case 2: if (pyy > 0) pyy--; break;
                            case 3: if (pyy < BOARD_WIDTH - 1) pyy++; break;
                        }
                        board[py * BOARD_WIDTH + px] = 0;
                        board[pyy * BOARD_WIDTH + pxx] = 1;
                        uint8_t cxx = cx;
                        uint8_t cyy = cy;
                        while ((pxx == cxx) && (pyy == cyy)) {
                            cxx = mixSeed(&seed1, &seed2) % BOARD_WIDTH;
                            cyy = mixSeed(&seed1, &seed2) % BOARD_WIDTH;
                        }
                        board[cyy * BOARD_WIDTH + cxx] = -1;
                        actions[queueIdx * ACTIONS + a] = a;
                        rewards[queueIdx * ACTIONS + a] = (pxx == cx) && (pyy == cy);
                        memcpy(nextStates + queueIdx * ACTIONS * BOARD_SIZE + a * BOARD_SIZE, board, BOARD_SIZE * sizeof(float));
                        board[cyy * BOARD_WIDTH + cxx] = 0;
                        board[pyy * BOARD_WIDTH + pxx] = 0;
                        board[py * BOARD_WIDTH + px] = 1;
                        board[cy * BOARD_WIDTH + cx] = -1;
                    }
                    queueIdx++;
                    board[py * 3 + px] = 0;
                    board[cy * 3 + cx] = 0;
                }
            }
        }
    }
    
    // print states
    for (uint32_t i = 0; i < NUM_FINAL_STATES; i++) {
        for (uint8_t dy = 0; dy < BOARD_WIDTH; dy++) {
            for (uint8_t dx = 0; dx < BOARD_WIDTH; dx++) {
                printf("%.0f ", states[i * BOARD_SIZE + dy * BOARD_WIDTH + dx]);
            }
            printf("\n");
        }
        printf("Action: %d\n", actions[i]);
        printf("Reward: %.0f\n", rewards[i]);
        for (uint8_t dy = 0; dy < BOARD_WIDTH; dy++) {
            for (uint8_t dx = 0; dx < BOARD_WIDTH; dx++) {
                printf("%.0f ", nextStates[i * BOARD_SIZE + dy * BOARD_WIDTH + dx]);
            }
            printf("\n");
        }
        printf("\n");
    }
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    Model model;
    Model frozenModel;
    initializeModel(&model, &seed1, &seed2);
    initializeModel(&frozenModel, &seed1, &seed2);
    
    for (uint32_t epoch = 0; epoch < (1 << 10); epoch++) {
        if (epoch % 8 == 0) {
            copyParams(&model, &frozenModel);
        }
        
        // outputGrad = rewards + DECAY * nextBestScore - output
        hipMemcpy(frozenModel.input, nextStates, BOARD_SIZE * NUM_FINAL_STATES * sizeof(float), hipMemcpyHostToDevice);
        forward(&handle, &frozenModel);
        float output[NUM_FINAL_STATES * ACTIONS];
        hipMemcpy(output, frozenModel.output, NUM_FINAL_STATES * ACTIONS * sizeof(float), hipMemcpyDeviceToHost);
        float nextBestScore[NUM_FINAL_STATES];
        for (uint32_t i = 0; i < NUM_FINAL_STATES; i++) {
            nextBestScore[i] = output[i * ACTIONS];
            for (uint8_t a = 1; a < ACTIONS; a++) {
                if (output[i * ACTIONS + a] > nextBestScore[i]) {
                    nextBestScore[i] = output[i * ACTIONS + a];
                }
            }
        }
        float outputGrad[NUM_FINAL_STATES]{};
        for (uint32_t i = 0; i < NUM_FINAL_STATES; i++) {
            outputGrad[i * ACTIONS + actions[i]] = rewards[i] + DECAY * nextBestScore[i] - output[i * ACTIONS + actions[i]];
        }
        
        float maxScore = 0;
        float minScore = 0;
        float avgScore = 0;
        for (uint32_t i = 0; i < NUM_FINAL_STATES; i++) {
            if (output[i * ACTIONS + actions[i]] > maxScore) {
                maxScore = output[i * ACTIONS + actions[i]];
            }
            if (output[i * ACTIONS + actions[i]] < minScore) {
                minScore = output[i * ACTIONS + actions[i]];
            }
            avgScore += output[i * ACTIONS + actions[i]];
        }
        avgScore /= NUM_FINAL_STATES;
        printf("Max: %f, Min: %f, Avg: %f\n", maxScore, minScore, avgScore);
        
        hipMemcpy(model.input, states, BOARD_SIZE * NUM_FINAL_STATES * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(model.outputGrad, outputGrad, NUM_FINAL_STATES * ACTIONS * sizeof(float), hipMemcpyHostToDevice);
        forward(&handle, &model);
        backward(&handle, &model);
    }

    return 0;
}