#include "Network.cuh"

// error handling for hipMalloc
void checkMallocError(hipError_t err) {
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        exit(1);
    }
}

int main(int argc, char** argv) {
    Noise noise;
    initNoise(&noise);
    
    Network net;
    uint32_t parameters[] = {3, 4, 1};
    uint32_t layers = sizeof(parameters) / sizeof(uint32_t) - 1;
    initNetwork(&net, parameters, layers, &noise, 0.1f, 1);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    float* testArr;
    // hipMalloc((void**)&testArr, 3 * sizeof(float));
    checkMallocError(hipMalloc((void**)&testArr, 3 * sizeof(float)));
    float testArrHost[] = {1.0f, 2.0f, 3.0f};
    hipMemcpy(testArr, testArrHost, 3 * sizeof(float), hipMemcpyHostToDevice);
    printf("Test array:\n");
    printTensor(testArr, 3, 1);
    
    for (uint32_t i = 0; i < 0; i++) {
        float testInput[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f};
        hipMemcpy(net.outputs[0], testInput, net.batchSize * net.parameters[0] * sizeof(float), hipMemcpyHostToDevice);
        printf("Input:\n");
        printTensor(net.outputs[0], parameters[0], net.batchSize);
        // forwardNoiseless(&handle, &net);
        
        // printTensor(net.outputs[net.layers], parameters[net.layers], net.batchSize);
        // float output[net.batchSize * net.parameters[net.layers]];
        // hipMemcpy(output, net.outputs[net.layers], sizeof(output), hipMemcpyDeviceToHost);
        
        // float testOutputGrad[2 * 1];
        // for (uint32_t i = 0; i < 2; i++) {
        //     testOutputGrad[i] = 1 - output[i];
        // }
        // hipMemcpy(net.outputGrads[net.layers], testOutputGrad, net.batchSize * net.parameters[net.layers] * sizeof(float), hipMemcpyHostToDevice);
        // backwardNoiseless(&handle, &net);
    }
    
    printParams(&net);
    
    return 0;
}